#include "hip/hip_runtime.h"

// TODO(nloomis): TOF docs
//
// reconstructs a hologram on a number of planes and find the magnitude and (steerable) gradient 
// at each depth. the maximum st_grad * (1 - mag) is recorded as a focus metric.
//
//
// Overview of steps in this code:
//
// load image into cpu memory (raw hologram, captured by camera); the image is
//   expected to be 2D (no RGB channels), but may be raw data from a Bayer-
//   patterned sensor
// transfer image from CPU to GPU memory
//
// take 2D FFT of the image --> fft2_of_hologram (size 2k x 2k, eg)
//
// for (z : reconstruction_depths) {
//	 compute a reconstruction kernel directly in the Fourier domain (same size
//     as fft2_of_hologram; 2k x 2k, eg)
//   multiply fft2_of_hologram * reconstruction_kernel
//   take Inverse 2D FFT of (fft2_of_hologram * reconstruction_kernel) 
//     --> reconstructed_image
//   
//   filter reconstructed_image with a gradient filter ("steerable derivative")
//     --> "gradient_magnitude_gpu" (steerable filter magnitude)
//   inverting reconstructed_image so that objects are bright and background
//     is dark -> dark_field_image
//   compute a focus metric, sim = gradient_magnitude_gpu * dark_field_image
//
//   compare the value of the focus metric for this z slice against the prior
//     best focus metric, keeping track of which z slice maximimizes the focus
//     metric
// }
//
// output from loop: map of where each pixel was best focused (same size as the
//   hologram, eg 2k x 2k); get both the depth and how in focus the pixel was
//
// transfer maps from GPU to CPU memory and back to Matlab
// 

#include "mex.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include "float.h"


// Converts a double to a float and stores the result back in the dest pointer.
// The float is downcast, discarding the least significant bits in the double.
//
// This is useful because CUDA prefers floats for its computations, while
// Matlab defaults to doubles.
void double_to_float(float* dest, double* source, const int n) {
	for (int i = 0; i < n; ++i)
		dest[i] = (float) source[i];
}


// Convert an unsigned char (or a uint8 in Matlab's type name scheme) to a
// float and store the result to dest. The char is converted to the nearest
// float. For example, 8 becomes 8.0. 
void uint8_to_float(float* dest, unsigned char* source, const int n) {
	for (int i = 0; i < n; ++i)
		dest[i] = (float) source[i];
}


// Performs integer division, but rounds up if the division is not exact.
// For example, 3 / 2 yields 1 for normal integer division, but roundUpDiv(3/2)
// returns 2.
int roundUpDiv(int num, int denom) {
	return (num / denom) + (!(num % denom) ? 0 : 1);
}


// Returns a * b, where a and b are complex-valued floats.
// This device kernel only runs on a CUDA device.
__device__ hipfftComplex complexMult32(hipfftComplex a, hipfftComplex b) {
	hipfftComplex tempResult;
	tempResult.x = a.x * b.x - a.y * b.y;
	tempResult.y = a.y * b.x + a.x * b.y;
	return tempResult;
}


// Returns a * sc, where a is a complex-valued float and sc is a scalar.
// This device kernel can only runs on a CUDA device.
__device__ hipfftComplex complexScale32(hipfftComplex a, float sc) {
	hipfftComplex tempResult;
	tempResult.x = a.x*sc;
	tempResult.y = a.y*sc;
	return tempResult;
}


// Returns the magnitude of the complex-valued float, a.
// This device kernel only runs on a CUDA device.
__device__ float complexMagnitude(float2 a) {
	return sqrtf( a.x * a.x + a.y * a.y );
}


#include "propagation_kernels.cu" //note: included here because the file
//references complexScale32 and complexMult32... I didn't want to edit 
//propagation_kernels.cu, and I'm too lazy right now to include headers.
//note that this is currently terrible coding, you should include headers.
// -nl, 2010



// CUDA threads and data arrangement
// =========================================
// 
// CUDA launches multiple compute threads, each expected to perform
// independently of the other threads. Threads are not all launched
// simultaneously, but are staged into smaller sets of threads known as blocks.
// The user can set the size of a block to optimize performance. The user can
// also set the shape of a block (for optimization or convenience). Block shapes
// can be multidimensional.
//
// A 2D block was used here for convenience to cover all the pixels of the 
// hologram. Note that other shapes, especially those which make use of memory
// concurrency, may actually give better performance.
//
// The number of blocks is also set by the user in order to best cover the
// data they want processed. If the size of the data is not evenly divisible
// by the size of a block, there will necessarily be some threads launched in
// some blocks which do not have corresponding data. It is the CUDA thread's
// responsibility to check for whether it has data it should access.
//
// Functions marked as __global__ are passed two additional arguments: the size
// of a block, and the index of the thread within the block. The thread is
// expected to use the size and index information to determine which data
// elements it needs to access. All the kernels here use a similar pattern:
//
//   static __global__ void DeviceKernel(T* dest, U* source,
//                                       additional args...,
//                                       const int M, const int N) {
//     // Find the pixel that this thread is responsible for computing, in
//     // two dimensional (x, y) coordinates.
//	   int xidx = threadIdx.x + blockDim.x * blockIdx.x;
//     int yidx = threadIdx.y + blockDim.y * blockIdx.y;
//     // Check that the pixel is within the known bounds of the source (and
//     // destination) data. If it is NOT within the bounds, no memory access is
//     // allowed and the thread exits early; segfaults and memory leaks can
//     // occur otherwise.
//     if (xidx < N && yidx < M) {
//       int idx = yidx * N + xidx;
//       // Compute something using data from source[idx] and storing back to
//       // dest[idx].
//     }
//   }
//


// Converts a float to a complex float, storing the result into dest. The real
// part of dest is copied from source, while the imaginary part is set to zero. The
// size of the source array is M rows and N columns.
static __global__ void float_to_complexKernel(float2* dest, float* source,
                                              const int M, const int N) {
	int xidx = threadIdx.x + blockDim.x * blockIdx.x;
	int yidx = threadIdx.y + blockDim.y * blockIdx.y;
	if (xidx < N && yidx < M){
		// Index of the data in the array.
		int idx = yidx * N + xidx;
		// dest.x is the real component.
		dest[idx].x = source[idx];
		// des.y is the imaginary component.
		dest[idx].y = 0;
	}
}


// In-place multiplication. The dest is a pointer to an array with size M rows
// and N columns; each value in dest is multipled by the constant scalar, sc.
static __global__ void scaleFloatKernel(float* dest, const float sc,
                                        const int M, const int N) {
	int xidx = threadIdx.x + blockDim.x * blockIdx.x;
	int yidx = threadIdx.y + blockDim.y * blockIdx.y;
	if (xidx < N && yidx < M){
		int idx = yidx * N + xidx;
		dest[idx] = dest[idx] * sc;
	}
}


// Copies the real part of the value in csrc and stores it to dest. Both csrc
// and dest have M rows and N columns.
static __global__ void copyRealKernel(float* dest, float2* csrc,
                                      const int M, const int N) {
	int xidx = threadIdx.x + blockDim.x * blockIdx.x;
	int yidx = threadIdx.y + blockDim.y * blockIdx.y;
	if (xidx < N && yidx < M){
		int idx = yidx * N + xidx;
		dest[idx] = csrc[idx].x;
	}
}

// Computes the magnitude of the complex-valued csrc pixel and stores it to dest.
// Both csrc and dest have M rows and N columns.
static __global__ void copyMagnitudeKernel(float* dest, float2* csrc,
                                           const int M, const int N) {
	int xidx = threadIdx.x + blockDim.x * blockIdx.x;
	int yidx = threadIdx.y + blockDim.y * blockIdx.y;
	if (xidx < N && yidx < M){
		int idx = yidx * N + xidx;
		dest[idx] = complexMagnitude(csrc[idx]);
	}
}


// TODO(nloomis): docs, with args...
//
// Compares a new floating point value against the last-known maximum to see
// if it is larger. If the new value is larger, the value is stored back as the
// new last-known maximum, and an index associated with where the maximum
// occurred is recorded to max_index.
//
// This kernel is used for finding which z-plane maximimzes a focus metric.
//
// Arguments:
//  max_metric : the current maximum value of the focus metric at each pixel;
//               if the new slice has a higher metric, the value replaces the
//               entry in max_metric.
//  index_of_max : an index recording the slice where the maximum metric
//               occurred. If the new slice's metric is larger than the
//               current maximum, the index associed with the new slice replaces
//               the entry in index_of_max.
//  slice_metric : a focus metric computed for a new reconstruction slice. If
//               slice_metric > max_metric, slice_metric replaces max_metric and
//               slice_index is stored to index_of_max.
//  slice_index : an index associated with the current slice, used to refer
//               back to the particular reconstruction.
//  M, N: max_metric, index_of_max, and slice_metric all have M rows and N
//               columns.
static __global__ void compareRealKernel(float* max_metric,
                                         short* index_of_max,
                                         float* slice_metric,
                                         const int slice_index,
                                         const int M,
                                         const int N) {
	int xidx = threadIdx.x + blockDim.x * blockIdx.x;
	int yidx = threadIdx.y + blockDim.y * blockIdx.y;
	if (xidx < N && yidx < M){
		int idx = yidx * N + xidx;
		float newreal = slice_metric[idx];
		float oldreal = max_float[idx];
		max_metric[idx] = newreal > oldreal ? newreal : oldreal;
		index_of_max[idx] = newreal > oldreal ?
			                (short) slice_index : index_of_max[idx];
	}
}


// Compares the focus metric of a reconstruction slice against the current
// maximum focus metric. If the pixel has a higher focus metric, the slice's
// focus metric, the slice's reconstruction intensity*, and the slice's
// reference index replace the values in max_metric, intensity_at_max_metric,
// and index_of_max_metric.
//
// This kernel is used to find the slice where the focus metric maximizes and
// the resulting intensity on that slice.
//
// *Reconstructions have both a real part and a complex part. The true intensity
// corresponds to the magnitude of the reconstruction. Because of the way
// propagation_kernels calculates the phase offset, the real component reflects
// most of the variation which occurs in intensity. It also requires fewer
// computations than the magnitude, and looks better to an observer. This
// function copies the real component into the intensity_at_max_metric array.
static __global__ void compareRealKernel(float* max_metric,
                                         short* index_of_max_metric,
                                         float* slice_metric,
                                         float* intensity_at_max_metric,
                                         float2* slice_intensity,
										 const int slice_index,
										 const int M,
										 const int N) {
	int xidx = threadIdx.x + blockDim.x * blockIdx.x;
	int yidx = threadIdx.y + blockDim.y * blockIdx.y;
	if (xidx < N && yidx < M){
		int idx = yidx * N + xidx;
		float newreal = slice_metric[idx];
		float oldreal = max_metric[idx];
		max_metric[idx] = (newreal > oldreal ? newreal : oldreal);
		index_of_max_metric[idx] = (newreal > oldreal ?
		                           (short) slice_index :
		                           index_of_max_metric[idx]);
		// If the focus metric is higher at this slice, record the intensity
		// (or, as a proxy, the real component in the x field) of this slice's
		// reconstruction; if not, keep the value the same.
		intensity_at_max_metric[idx] = (newreal > oldreal ?
		                               slice_intensity[idx].x :
		                               intensity_at_max_metric[idx]);
	}
}

 
// Compares the real value of a reconstruction slice against the smallest known
// real value of previous reconstructions. If the slice's value is smaller, the
// slice's value and the index of the slice overwrite the entries in the
// minimum_real and index_at_min arrays. The arrays all have M rows and N
// columns.
static __global__ void compareReal2Kernel(float* minimum_real,
                                          short* index_at_min,
                                          float2* reconstructed_slice,
										  const int slice_index,
										  const int M, const int N) {
	int xidx = threadIdx.x + blockDim.x * blockIdx.x;
	int yidx = threadIdx.y + blockDim.y * blockIdx.y;
	if (xidx < N && yidx < M){
		int idx = yidx * N + xidx;
		float slice_real_value = reconstructed_slice[idx].x;
		float current_minimum = minimum_real[idx];
		minimum_real[idx] = (slice_real_value < current_minimum ?
		                     slice_real_value :
		                     oldreal);
		index_at_min[idx] = (slice_real_value < current_minimum ?
		                    (short) slice_index :
		                    index_at_min[idx]);
	}
}


// In-place computation of the magnitude of a complex number. The arrray of
// complex numbers, source, has M rows and N columns. The magnitude of source is
// stored back to the real part of source, and the imaginary part is set to zero.
static __global__ void computeMagKernel(float2* source,
                                        const int M,
                                        const int N) {
	int xidx = threadIdx.x + blockDim.x * blockIdx.x;
	int yidx = threadIdx.y + blockDim.y * blockIdx.y;
	if (xidx < N && yidx < M) {
		int idx = yidx * N + xidx;
		source[idx].x = complexMagnitude(source[idx]);
		source[idx].y = 0;
	}
}

// Multiplies arrays of complex numbers in source1 and source2 and stores the
// result to dest. The arrays all have M rows and N columns.
static __global__ void ComplexMultiplicationKernel(float2* dest,
                                                   float2* source1,
                                                   float2* source2,
                                                   const int M,
                                                   const int N) {
	int xidx = threadIdx.x + blockDim.x * blockIdx.x;
	int yidx = threadIdx.y + blockDim.y * blockIdx.y;
	if (xidx < N && yidx < M) {
		int idx = yidx * N + xidx;
		dest[idx] = complexMult32(source1[idx], source2[idx]);
	}
}


// Returns the total magnitude of a gradient filter, computed as the L2 norm
// of the x- and y-direction gradients. The x- and y-direction gradients are
// assumed to have been computed on a complex-valued field, and the magnitudes
// of their complex responses are calculated first before finding the net
// response.
//
// The gradient magnitude is computed as:
//  x_mag = sqrt(real(x)^2 + imag(x)^2)
//  y_mag = sqrt(real(y)^2 + imag(y)^2)
//  grad_mag = sqrt(x_mag^2 + y_mag^2)
//
// gradient_magnitude: resulting magnitude of the image gradient
// gradient_in_x: the x-direction response of a gradient filter applied to the
//             complex-valued image
// gradient_in_y: the y-direction response of a gradient filter applied to the
//             complex-valued image
// M, N: number of rows and columns in each of the data arrays
//
static __global__ void GradientMagnitudeKernel(float* gradient_magnitude,
                                              float2* gradient_in_x,
                                              float2* gradient_in_y, 
	                                          const int M,
	                                          const int N) {
	int xidx = threadIdx.x + blockDim.x * blockIdx.x;
	int yidx = threadIdx.y + blockDim.y * blockIdx.y;
	if (xidx < N && yidx < M) {
		int idx = yidx * N + xidx;
		float x_magnitue = complexMagnitude(gradient_in_x[idx]);
		float y_magnitude = complexMagnitude(gradient_in_y[idx]);
		gradient_magnitude[idx] = sqrtf(x_magnitude * xmagnitude +
		                                y_magnitude * y_magnitude);
	}
}	


// SIM is the product of the gradient ("S") and image magnitude ("IM"). The
// metric considers sharp edgegs which are dark to be favorable. Since "dark" 
// has a smaller numerical value, the image brightness, or local magnitude, is
// subtracted from the expected maximum intensity so that 
//
//   (maximum_intensity - image_magnitude)
//
// gives a large value for favorable dark pixels. The SIM is then
//
//   SIM = gradient_magnitude * (maximum_intensity - image_magnitude).
//
// sim: output array of the SIM focus metric
// gradient_magnitude: magnitude of the a gradient filter which has been applied
//           to the complex-valued reconstructed image
// image_magnitude: magnitude of a single slice's reconstructed image
// maximum_intensity: maximum intensity expected over all slices; should be a
//           constant across slices
// M: number of rows in sim, gradient_magnitude, and image_magnitude
// N: number of cols in sim, gradient_magnitude, and image_magnitude
static __global__ void computeSIMkernel(float* sim, 
	                                    float* gradient_magnitude,
                                        float* image_magnitude,
                                        const float maximum_intensity, 
										const int M, const int N) {
	int xidx = threadIdx.x + blockDim.x * blockIdx.x;
	int yidx = threadIdx.y + blockDim.y * blockIdx.y;
	if (xidx < N && yidx < M) {
		int idx = yidx * N + xidx;
		sim[idx] = gradient_magnitude[idx] *
		          (maximum_intensity - image_magnitude[idx]);
	}
}


// In-place addition of shorts. The kernel is used to add value to all elements
// of dest. The dest array has M rows and N columns.
static __global__ void AddShortsKernel(short* dest, const short value,
                                      const int M, const int N) {
	int xidx = threadIdx.x + blockDim.x * blockIdx.x;
	int yidx = threadIdx.y + blockDim.y * blockIdx.y;
	if (xidx < N && yidx < M){
		dest[yidx * N + xidx] += value;
	}
}



// TODO(nloomis): docs
//NB: the source and destination are the same location!
//TODO: this might be a good use for atomic operators.
static __global__ void PowerKernel(float2* fsrc,
                                   const int N, const int M,
								   const float power_order,
								   const float power_filter_offset,
								   const float sigma) {
	int xidx = threadIdx.x + blockDim.x * blockIdx.x; //pixel index that this thread deals with
	int yidx = threadIdx.y + blockDim.y * blockIdx.y;
	if (xidx < N && yidx < M){
		int idx = yidx * N + xidx;

		//get the FFTW-shifted u,v coordinates (in numbers of samples)
		//option 4) use normalized freqs (just the magnitude) - note that uidx->u here in the nomenclature.
		int halfx = N / 2;
		int halfy = M / 2;
		float u = (xidx >= halfx ? (float) (N - xidx) / (float) halfx :
		                           (float) xidx / (float) halfx);
		float v = (yidx >= halfy ? (float) (M - yidx) / (float) halfy :
		                           (float)yidx / (float) halfy);

		float2 value = fsrc[idx]; //the value to work with

		//do some operation on value using u,v
		//here, i'm using a power filter to keep the lower freqs but kill the high freqs.
		float fkeep = __expf(-( __powf(u, power_order) +
		                        __powf(v, power_order)) /
		                      sigma);
		value.x = value.x * fkeep;
		value.y = value.y * fkeep; 
		//the power_filter_offset is the fraction to keep no matter what; only (1-power_filter_offset) of the value
		//is allowed to be changed.

		// TODO(nloomis): where is power_filter_offset used?!?

		//store the result back to the destination
		fsrc[idx] = value;
	}
}

// Computes the convolution kernel for a steerable filter in the x-direction. A
// steerable filter has two important characteristics:
//  1) the filter includes some smoothing, so the effects of random noise are
//     reduced, and
//  2) the gradient magnitude is independent of the orientation of an edge in
//     an image.
// 
// TODO(nloomis): finish docs.
// note: use float2 so that fft2 can be taken immediately
static __global__ void fillSfiltxKernel(float2* gradient_filter_in_x, const float sigma, 
                                        const int N, const int M) {
	int xidx = threadIdx.x + blockDim.x * blockIdx.x;
	int yidx = threadIdx.y + blockDim.y * blockIdx.y;
	if (xidx < N && yidx < M){
        int idx = yidx * N + xidx;
        
		int halfx = N / 2;
		int halfy = M / 2;
		float u = (xidx >= halfx ? -(float)(N - xidx) : (float) xidx);
		float v = (yidx >= halfy ? -(float)(M - yidx) : (float) yidx);
		
		//set the value
		gradient_filter_in_x[idx].x = -2.0f * u * __expf(-(u * u + v * v) /
		                             (2.0f * sigma * sigma));
        // The imaginary component is zero.
        gradient_filter_in_x[idx].y = 0;

	}
}


// Computes the convolution kernel for a steerable filter in the y-direction.
// See documentation for fillSfiltxKernel above.
//
static __global__ void fillSfiltyKernel(float2* gradient_filter_in_y, const float sigma, 
                                        const int xpix, const int ypix) {
	int xidx = threadIdx.x + blockDim.x * blockIdx.x;
	int yidx = threadIdx.y + blockDim.y * blockIdx.y;
	if (xidx < xpix && yidx < ypix){
		int halfx = xpix / 2;
		int halfy = ypix / 2;
		float u = (xidx >= halfx ? -(float)(xpix - xidx) : (float)xidx);
		float v = (yidx >= halfy ? -(float)(ypix - yidx) : (float)yidx);

		int idx = yidx * ypix + xidx; //the linear index of the data
		
		//set the value
		gradient_filter_in_y[idx].x = -2.0f * v * __expf(-(u * u + v * v) /
			            (2.0f * sigma * sigma));
        // The imaginary component is zero.
        gradient_filter_in_y[idx].y = 0;
	}
}



void reconstruct(float* focus_metric, short* index_of_max_focus_metric, float* min_intensity_cpu, short* index_of_min_intensity, float* hologram_cpu, 
					 double *z, const int nz, const int M, const int N, 
					 const double wavelength, const double pixel_size,
					 const float power_filter_order, const float power_filter_offset,
					 const float steerable_filter_sigma, const float maximum_intensity,
					 float* intensity_at_best_focus_cpu, bool rintmode, bool record_intensity_at_best_focus) {
	float* hologram_gpu, *min_intensity_gpu, *simmaxdev, *gradient_magnitude_gpu, *image_magnitude, *sim, *intensity_at_best_focus_gpu;
	short* index_of_min_intensity_gpu, *index_of_max_focus_metric_gpu;
	float2* chologram_gpu, *reconstruction_gpu, *gradient_filter_in_x, *gradient_filter_in_y, *sx, *sy;
	float adudu, cutoff2, fftscale;
	float2 offsetPhaseExp;
	float sigma;

    // A block is the number of threads which are launched at each round. A size
    // of 16x16 is used to step through the image 256 threads at a time.
	dim3 myblock(16, 16);
    // TODO(nloomis): doc
	dim3 mygrid( roundUpDiv(N,16), roundUpDiv(M,16) );

	mexPrintf("Uploading hologram\n");

	//upload the hologram and convert it to a complex-valued matrix
	hipMalloc( (void**)&hologram_gpu, sizeof(float)*M*N);
	hipMalloc( (void**)&chologram_gpu, sizeof(float2)*M*N);
	hipMemcpy( hologram_gpu, hologram_cpu, sizeof(float)*M*N, hipMemcpyHostToDevice);
	float_to_complexKernel<<<mygrid, myblock>>>(chologram_gpu, hologram_gpu, M, N);
	hipFree(hologram_gpu);

	mexPrintf("Creating memory on GPU\n");

	//create variables to track the max focus_metric, min int
	hipMalloc( (void**)&simmaxdev, sizeof(float)*M*N);
	hipMalloc( (void**)&index_of_max_focus_metric_gpu, sizeof(short)*M*N);
	hipMemset(index_of_max_focus_metric_gpu, 0, sizeof(short)*M*N);
	//fillFloatKernel<<<mygrid, myblock>>>(min_intensity_gpu, FLT_MIN, M, N); //don't need this
	if (rintmode){
		hipMalloc( (void**)&min_intensity_gpu, sizeof(float)*M*N);
		hipMalloc( (void**)&index_of_min_intensity_gpu, sizeof(short)*M*N);
		hipMemset(index_of_min_intensity_gpu, 0, sizeof(short)*M*N);
	}

	mexPrintf("Creating FFT plan\n");

	//create an FFT plan
	hipfftHandle plan;
	hipfftPlan2d(&plan, N, M, HIPFFT_C2C);

	mexPrintf("Executing FFT\n");
	
	//compute the (power-modulated) spectra, which will be used for each reconstruction step
	//take the FFT of the hologram data
	hipfftExecC2C(plan, chologram_gpu, chologram_gpu, HIPFFT_FORWARD);

	mexPrintf("Applying power filter\n");

	//apply the power kernel filter
	if (power_filter_order > 0) {
		//mexPrintf("Applying power kernel\n");
		sigma = -1.0f/log(power_filter_offset);
		PowerKernel<<<mygrid, myblock>>>(chologram_gpu, N, M,
			power_filter_order, power_filter_offset, sigma);
	}

	mexPrintf("Computing steerable filter\n");

	//compute the steerable filter, used for each reconstruction step
	hipMalloc((void**) &gradient_filter_in_x, sizeof(float2) * M * N);
	hipMalloc((void**) &gradient_filter_in_y, sizeof(float2) * M * N);
	fillSfiltxKernel<<<mygrid, myblock>>>(gradient_filter_in_x, steerable_filter_sigma, M, N);
	fillSfiltyKernel<<<mygrid, myblock>>>(gradient_filter_in_y, steerable_filter_sigma, M, N);
	hipfftExecC2C(plan, gradient_filter_in_x, gradient_filter_in_x, HIPFFT_FORWARD); //compute the freq-domain version of the filter
	hipfftExecC2C(plan, gradient_filter_in_y, gradient_filter_in_y, HIPFFT_FORWARD);
	//..and reserve memory for the resulting sx, sy gradients
	hipMalloc((void**) &sx, sizeof(float2) * M * N);
	hipMalloc((void**) &sy, sizeof(float2) * M * N);
	hipMalloc((void**) &gradient_magnitude_gpu, sizeof(float) * M * N);



	//reserve memory for the reconstruction
	hipMalloc( (void**)&reconstruction_gpu, sizeof(float2) * M * N);


	//reserve memory for the focus metric components
	hipMalloc( (void**)&image_magnitude, sizeof(float) * M * N);
	hipMalloc( (void**)&sim, sizeof(float) * M * N);
	if (record_intensity_at_best_focus)
		hipMalloc( (void**)&intensity_at_best_focus_gpu, sizeof(float) * M * N);

	//fillFloatKernel<<<mygrid, myblock>>>(image_magnitude, 1.0f, M, N); //don't need this

	mexPrintf("Computing reconstructions\n");

	//reconstruct the hologram at each depth in z, compare it to the previous
	//minimum intensity
	for (int i=0; i<nz; i++){
		//copy the fft spectrum to a variable that's temporary for each plane
		hipMemcpy(reconstruction_gpu, chologram_gpu, sizeof(float2)*M*N, hipMemcpyDeviceToDevice);

		//set the propagation parameters
		setPropagationParams( (float)(z[i]), (float)wavelength, (float)pixel_size, M, N, 
			&adudu, &cutoff2, &offsetPhaseExp, &fftscale, false);

		//apply the propagation kernel (full... could speed this up by ignoring cutoff)
		fresnelKernel<<<mygrid, myblock>>>(reconstruction_gpu, M, N, adudu, 
			cutoff2, offsetPhaseExp, fftscale);

		//take the inverse transform: get an image at the plane.
		hipfftExecC2C(plan, reconstruction_gpu, reconstruction_gpu, HIPFFT_BACKWARD);

		//compute the magnitude
		computeMagKernel<<<mygrid, myblock>>>(reconstruction_gpu, M, N);

		//compute the steerable gradient responses in the x and y directions
		hipMemcpy( sx, reconstruction_gpu, sizeof(float2) * M * N, hipMemcpyDeviceToDevice); //copy the magnitude info
		hipfftExecC2C(plan, sx, sx, HIPFFT_FORWARD); //the spectrum of the magnitude
		hipMemcpy( sy, sx, sizeof(float2) * M * N, hipMemcpyDeviceToDevice); //copy the spectrum information
		//apply the sfilt kernels in the x and y directions
		ComplexMultiplicationKernel<<<mygrid, myblock>>>(sx, sx, gradient_filter_in_x, M, N);
		ComplexMultiplicationKernel<<<mygrid, myblock>>>(sy, sy, gradient_filter_in_y, M, N);
		hipfftExecC2C(plan, sx, sx, HIPFFT_BACKWARD);
		hipfftExecC2C(plan, sy, sy, HIPFFT_BACKWARD);
		//finally: compute the magnitude and orientation
		GradientMagnitudeKernel<<<mygrid,myblock>>>(gradient_magnitude_gpu, sx, sy, M, N);
		scaleFloatKernel<<<mygrid,myblock>>>(gradient_magnitude_gpu, 1.0f / ((float) (M * M) * (float) (N * N)), M, N);

		//find the local min intensity (for focus_metric)
		//localMinKernel<<<mygrid, myblock>>>(image_magnitude, reconstruction_gpu, M, N);
		copyRealKernel<<<mygrid, myblock>>>(image_magnitude, reconstruction_gpu, M, N);


		//compute the SIM metric
		computeSIMkernel<<<mygrid, myblock>>>(sim, gradient_magnitude_gpu, image_magnitude, maximum_intensity, M, N);
		

		if (rintmode){
			//compare the current intensity to the previous minimum
			if (i == 0) 
				copyRealKernel<<<mygrid,myblock>>>(min_intensity_gpu, reconstruction_gpu, M, N);
			else 
				compareReal2Kernel<<<mygrid, myblock>>>(min_intensity_gpu, index_of_min_intensity_gpu, reconstruction_gpu, i, M, N);
		}

		//compare the current SIM to the previous SIMs
		if (i == 0) {
			hipMemcpy(simmaxdev, sim, sizeof(float)*M*N, hipMemcpyDeviceToDevice);
			if (record_intensity_at_best_focus)
				hipMemcpy(intensity_at_best_focus_gpu, min_intensity_gpu, sizeof(float)*M*N, hipMemcpyDeviceToDevice);
		}
		else {
			if (!record_intensity_at_best_focus)
				compareRealKernel<<<mygrid, myblock>>>(
                    simmaxdev, 
                    index_of_max_focus_metric_gpu,
                    sim,
                    i, 
                    M, N);
			else //record_intensity_at_best_focus == true
				compareRealKernel<<<mygrid, myblock>>>(
                    simmaxdev,
                    index_of_max_focus_metric_gpu, 
                    sim, 
                    intensity_at_best_focus_gpu, 
                    reconstruction_gpu, 
                    i, 
                    M, N);
		}


	}

	// C-style indexing uses 0 as its first element, while Matlab uses 1 for its
    // first element. Add a 1 to all the slice indices recorded so that the
    // returned indices are directly usable in Matlab.
	if (rintmode)
		AddShortsKernel<<<mygrid, myblock>>>(index_of_min_intensity_gpu, 1, M, N);
	AddShortsKernel<<<mygrid, myblock>>>(
        index_of_max_focus_metric_gpu, 1, M, N);

    // Copy results from the GPU back to the CPU memory.
	mexPrintf("Copying results back to host\n");
	hipMemcpy(focus_metric, simmaxdev, sizeof(float) * M * N, hipMemcpyDeviceToHost);
	hipMemcpy(index_of_max_focus_metric, index_of_max_focus_metric_gpu, sizeof(short) * M * N, hipMemcpyDeviceToHost);
	if (rintmode) {
		hipMemcpy(min_intensity_cpu, min_intensity_gpu, sizeof(float) * M * N, hipMemcpyDeviceToHost);
		hipMemcpy(index_of_min_intensity, index_of_min_intensity_gpu, sizeof(short) * M * N, hipMemcpyDeviceToHost);
	}
	if (record_intensity_at_best_focus)
		hipMemcpy(intensity_at_best_focus_cpu, intensity_at_best_focus_gpu, sizeof(float) * M * N, hipMemcpyDeviceToHost);

	mexPrintf("Cleaning up memory and plans\n");

	//clean up!
	hipfftDestroy(plan);
	hipFree(chologram_gpu);
	hipFree(reconstruction_gpu);
	if (rintmode){
		hipFree(min_intensity_gpu);
		hipFree(index_of_min_intensity_gpu);
	}
	hipFree(gradient_filter_in_x);
	hipFree(gradient_filter_in_y);
	hipFree(sx);
	hipFree(sy);
	hipFree(gradient_magnitude_gpu);
	hipFree(sim);
	hipFree(image_magnitude);
	hipFree(index_of_max_focus_metric_gpu);
	hipFree(simmaxdev);
	if (record_intensity_at_best_focus)
		hipFree(intensity_at_best_focus_gpu);
}



// [focus_metric, index_of_max_focus_metric, min_intensity_cpu, index_of_min_intensity, R_at_SIM] = maxSIMfm(img, z, wavelength, pixel_size, ...
//           powerKernOrder, powerKernOffset, sigma, maximum_intensity);
void mexFunction( int nlhs, mxArray* plhs[],
                  int nrhs, const mxArray* prhs[]) {

    // Pointer for the hologram data.
	float* hologram_cpu;
    // Minimum reconstruction intensity.
    float* min_intensity_cpu;
    // Stores a metric for how well each hologram pixel can be focused.
    float* focus_metric;
    float* intensity_at_best_focus_cpu = NULL;
    // Pointer for source hologram data, if it is passed in as a double.
	double* dsrc;
    // Pointer for source hologram data, if it is passed in as an unsigned char.
	unsigned char* isrc;
	short* index_of_min_intensity;
    short* index_of_max_focus_metric;
	double* z;
    // Wavelength of the illumination light in meters.
    double wavelength = 658e-9;
    // Size of a camera pixel in meters.
    double pixel_size = 9e-6;
	int M, N, nz;
	float power_filter_order = -1, power_filter_offset = .001, sigma = 2, maximum_intensity = 1000;
	bool rintmode, record_intensity_at_best_focus;

	if (nrhs<2)
		mexErrMsgTxt("maxSIMfm(image, z, [wavelength], [pixel_size]) requires at least two arguments.");

	//get the size of the hologram
	M = mxGetN(prhs[0]); //number of y-direction pixels
	N = mxGetM(prhs[0]); //number of x-direction pixels
	//NB: I'm mixing M, N so that they align with the Matlab imagesc plots.
	//M is the y-direction , N is the x-direction.
	//this is because Matlab stores its data reading down the rows (row-major), while
	//C is column-wise ordering (column-major)
	
	//retrieve the depths at which to do reconstructions
	z = mxGetPr(prhs[1]);
	nz = mxGetM(prhs[1]) * mxGetN(prhs[1]);
	
	//mexPrintf("M: %i, N: %i, nz: %i\n", M, N, nz);

	//get optional parameters; use defaults otherwise
	if (nrhs>=3)
		if (!mxIsEmpty(prhs[2]))
			wavelength = mxGetScalar(prhs[2]);
	if (nrhs>=4)
		if (!mxIsEmpty(prhs[3]))
			pixel_size = mxGetScalar(prhs[3]);
	if (nrhs>=5)
		if (!mxIsEmpty(prhs[4]))
			power_filter_order = (float) mxGetScalar(prhs[4]);
	if (nrhs>=6)
		if (!mxIsEmpty(prhs[5]))
			power_filter_offset = (float) mxGetScalar(prhs[5]);
	if (nrhs>=7)
		if (!mxIsEmpty(prhs[6]))
			sigma = (float) mxGetScalar(prhs[6]);
	if (nrhs>=8)
		if (!mxIsEmpty(prhs[7]))
			maximum_intensity = (float) mxGetScalar(prhs[7]);
			
    // Retrieve the pointer to the hologram data. If the data type is not float,
    // it needs to be converted for to float for efficient computations on the
    // GPU.
	mxClassID imgclass = mxGetClassID(prhs[0]);
	switch (imgclass) {
		case mxSINGLE_CLASS:
		    // If the hologram was passed in as a float, copy its pointer.
			hologram_cpu = (float*) mxGetPr(prhs[0]);
			break;
		case mxDOUBLE_CLASS:
		    // If the hologram was passed in as a double, it needs to be
		    // down-cast to float.
			dsrc = mxGetPr(prhs[0]);
			hologram_cpu = (float*) mxMalloc(sizeof(float) * M * N);
			double_to_float(hologram_cpu, dsrc, M * N);
			break;
		case mxUINT8_CLASS:
		    // If the hologram was passsed in as an unsigned char (what Matlab
		    // calls "uint8"), it needs to be converted to float.
			isrc = (unsigned char*) mxGetPr(prhs[0]);
			hologram_cpu = (float*) mxMalloc(sizeof(float) * M * N);
			uint8_to_float(hologram_cpu, isrc, M * N);
			break;
		default:
			mexErrMsgTxt("Input image needs to be single, double, or uint8.");
			break;
	}
	
	//create outputs
	
	if (nlhs > 2)
		rintmode = true;
	else
		rintmode = false;
	if (nlhs > 4)
		record_intensity_at_best_focus = true;
	else
		record_intensity_at_best_focus = false;

	plhs[0] = mxCreateNumericMatrix(N, M, mxSINGLE_CLASS, mxREAL); //the order is reversed here
	plhs[1] = mxCreateNumericMatrix(N, M, mxINT16_CLASS, mxREAL); //from what you'd expect... row-major vs column-major.
	focus_metric = (float*) mxGetPr(plhs[0]);
	index_of_max_focus_metric = (short*) mxGetPr(plhs[1]);

	if (rintmode) {
		plhs[2] = mxCreateNumericMatrix(N, M, mxSINGLE_CLASS, mxREAL);
		plhs[3] = mxCreateNumericMatrix(N, M, mxINT16_CLASS, mxREAL);
		min_intensity_cpu = (float*) mxGetPr(plhs[2]); //single
		index_of_min_intensity = (short*) mxGetPr(plhs[3]); //int16
	}

	if (record_intensity_at_best_focus){
		plhs[4] = mxCreateNumericMatrix(N, M, mxSINGLE_CLASS, mxREAL);
		intensity_at_best_focus_cpu  = (float*) mxGetPr(plhs[4]);
	}

	//reconstruct hologram and find mins
	reconstruct(SIM, index_of_max_focus_metric, min_intensity_cpu, index_of_min_intensity, hologram_cpu, z, nz, M, N, wavelength, pixel_size, 
		power_filter_order, power_filter_offset, sigma, maximum_intensity, intensity_at_best_focus_cpu, rintmode, record_intensity_at_best_focus);

	// If the hologram was originally passed in as a double or unsigned chars,
	// hologram_cpu would have been malloc'd. Free it now, before exiting.
	if ( imgclass == mxDOUBLE_CLASS || imgclass == mxUINT8_CLASS )
		mxFree(hologram_cpu);
}
