#include "hip/hip_runtime.h"

//reconstructs a hologram on a number of planes and find the magnitude and (steerable) gradient 
//at each depth. the maximum st_grad * (1 - mag) is recorded as a focus metric.



#include "mex.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include "float.h"





void double_to_float(float *dst, double *src, const int n)
{
	for (int i=0; i<n; i++)
		dst[i] = (float)src[i];
}




void uint8_to_float(float *dst, unsigned char *src, const int n)
{
	for (int i=0; i<n; i++)
		dst[i] = (float)src[i];
}




//integer division, but forces a round up if there are leftovers (ie, 3/2 = 1, but roundUpDiv(3/2) = 2)
int roundUpDiv(int num, int denom){
	return (num/denom) + (!(num%denom)? 0 : 1);
}




__device__ hipfftComplex complexMult32(hipfftComplex a, hipfftComplex b)
{
	hipfftComplex tempResult;
	tempResult.x = a.x * b.x - a.y * b.y;
	tempResult.y = a.y * b.x + a.x * b.y;
	//tempResult.x = __fmul_rn(a.x,b.x) - __fmul_rn(a.y,b.y);
	//tempResult.y = __fadd_rn( __fmul_rn(a.y,b.x) , __fmul_rn(a.x,b.y));
	return tempResult;
}




//multiply a complex value by a scale factor
__device__ hipfftComplex complexScale32(hipfftComplex a, float sc)
{
	hipfftComplex tempResult;
	tempResult.x = a.x*sc;
	tempResult.y = a.y*sc;
	//tempResult.x = __fmul_rn(a.x,sc);
	//tempResult.y = __fmul_rn(a.y,sc);
	return tempResult;
}



__device__ float complexMagnitude(float2 a)
{
	return sqrtf( a.x*a.x + a.y*a.y );
	//return __fsqrt_rn( __fmul_rn(a.x,a.x) + __fmul_rn(a.y,a.y) );
	//return __fsqrt_rn( a.x*a.x + a.y*a.y );
}


#include "propagation_kernels.cu" //note: included here because the file
//references complexScale32 and complexMult32... I didn't want to edit 
//propagation_kernels.cu, and I'm too lazy right now to include headers.
//note that this is currently terrible coding, you should include headers.
//





static __global__ void float_to_complexKernel(float2 *dst, float *src, const int M, const int N)
{
	int xidx = threadIdx.x + blockDim.x*blockIdx.x;
	int yidx = threadIdx.y + blockDim.y*blockIdx.y;
	if (xidx<N && yidx<M){
		int idx = yidx*N + xidx;
		dst[idx].x = src[idx];
		dst[idx].y = 0;
	}
}



/*
static __global__ void fillFloatKernel(float *dst, const float value, 
									   const int M, const int N)
{
	int xidx = threadIdx.x + blockDim.x*blockIdx.x;
	int yidx = threadIdx.y + blockDim.y*blockIdx.y;
	if (xidx<N && yidx<M){
		dst[yidx*N+xidx] = value;
	}
}
*/


static __global__ void scaleFloatKernel(float *dst, const float sc, const int M, const int N)
{
	int xidx = threadIdx.x + blockDim.x*blockIdx.x;
	int yidx = threadIdx.y + blockDim.y*blockIdx.y;
	if (xidx<N && yidx<M){
		int idx = yidx*N + xidx;
		dst[idx] = dst[idx]*sc;
	}
}



static __global__ void copyRealKernel(float *dst, float2 *csrc, const int M, const int N)
{
	int xidx = threadIdx.x + blockDim.x*blockIdx.x;
	int yidx = threadIdx.y + blockDim.y*blockIdx.y;
	if (xidx<N && yidx<M){
		int idx = yidx*N+xidx;
		dst[idx] = csrc[idx].x;
	}
}


static __global__ void copyMagnitudeKernel(float *dst, float2 *csrc, const int M, const int N)
{
	int xidx = threadIdx.x + blockDim.x*blockIdx.x;
	int yidx = threadIdx.y + blockDim.y*blockIdx.y;
	if (xidx<N && yidx<M){
		int idx = yidx*N+xidx;
		dst[idx] = complexMagnitude(csrc[idx]);
	}
}


//finds the larger of the two
static __global__ void compareRealKernel(float *rintdev, short *ridxdev, float *recdev,
										  const int i, const int M, const int N)
{
	int xidx = threadIdx.x + blockDim.x*blockIdx.x;
	int yidx = threadIdx.y + blockDim.y*blockIdx.y;
	if (xidx<N && yidx<M){
		int idx = yidx*N+xidx;
		float newreal = recdev[idx];
		float oldreal = rintdev[idx];
		rintdev[idx] = (newreal > oldreal ? newreal : oldreal);
		ridxdev[idx] = (newreal > oldreal ? (short)i : ridxdev[idx]);
	}
}


//finds the larger of the two; this is the overloaded version that stores the intensity at the max sim as well.
static __global__ void compareRealKernel(float *sim, short *sidxdev, float *simdev, float *rintdev, float2 *recdev,
										  const int i, const int M, const int N)
{
	int xidx = threadIdx.x + blockDim.x*blockIdx.x;
	int yidx = threadIdx.y + blockDim.y*blockIdx.y;
	if (xidx<N && yidx<M){
		int idx = yidx*N+xidx;
		float newreal = simdev[idx];
		float oldreal = sim[idx];
		sim[idx] = (newreal > oldreal ? newreal : oldreal);
		sidxdev[idx] = (newreal > oldreal ? (short)i : sidxdev[idx]);
		rintdev[idx] = (newreal > oldreal ? recdev[idx].x : rintdev[idx]);
	}
}




//finds the smaller of the two
static __global__ void compareReal2Kernel(float *rintdev, short *ridxdev, float2 *recdev,
										  const int i, const int M, const int N)
{
	int xidx = threadIdx.x + blockDim.x*blockIdx.x;
	int yidx = threadIdx.y + blockDim.y*blockIdx.y;
	if (xidx<N && yidx<M){
		int idx = yidx*N+xidx;
		float newreal = recdev[idx].x;
		float oldreal = rintdev[idx];
		rintdev[idx] = (newreal < oldreal ? newreal : oldreal);
		ridxdev[idx] = (newreal < oldreal ? (short)i : ridxdev[idx]);
	}
}


/*
static __global__ void compareIntensities(float *rintdev, short *ridxdev, float2 *recdev,
										  const int i, const int M, const int N)
{
	int xidx = threadIdx.x + blockDim.x*blockIdx.x;
	int yidx = threadIdx.y + blockDim.y*blockIdx.y;
	if (xidx<N && yidx<M){
		int idx = yidx*N+xidx;
		float newmag = complexMagnitude( recdev[idx] );
		float oldmag = rintdev[idx];
		rintdev[idx] = (newmag < oldmag ? newmag : oldmag);
		ridxdev[idx] = (newmag < oldmag ? (short)i : ridxdev[idx]);
	}
}
*/

static __global__ void computeMagKernel(float2 *src, const int M, const int N)
{
	int xidx = threadIdx.x + blockDim.x*blockIdx.x;
	int yidx = threadIdx.y + blockDim.y*blockIdx.y;
	if (xidx<N && yidx<M) {
		int idx = yidx*N+xidx;
		src[idx].x = complexMagnitude(src[idx]);
		src[idx].y = 0;
	}
}




static __global__ void complexMultKernel(float2 *dst, float2 *src1, float2 *src2, const int M, const int N)
{
	int xidx = threadIdx.x + blockDim.x*blockIdx.x;
	int yidx = threadIdx.y + blockDim.y*blockIdx.y;
	if (xidx<N && yidx<M) {
		int idx = yidx*N+xidx;
		dst[idx] = complexMult32(src1[idx], src2[idx]);
	}
}



/*
static __global__ void applySyKernel(float2 *src, float2 *sfiltx, const int M, const int N)
{
	int xidx = threadIdx.x + blockDim.x*blockIdx.x;
	int yidx = threadIdx.y + blockDim.y*blockIdx.y;
	if (xidx<N && yidx<M) {
		int idx = yidx*N+xidx;
		int idx_t = xidx*M + yidx; //TODO: check this if M!=N
		src[idx] = complexMult32(src[idx], sfiltx[idx_t]);
	}
}
*/


static __global__ void sfiltMagKernel(float *sfiltmag, float2 *sx, float2 *sy, const int M, const int N){
	int xidx = threadIdx.x + blockDim.x*blockIdx.x;
	int yidx = threadIdx.y + blockDim.y*blockIdx.y;
	if (xidx<N && yidx<M) {
		int idx = yidx*N + xidx;
		float sxmag = complexMagnitude(sx[idx]);
		float symag = complexMagnitude(sy[idx]);
		sfiltmag[idx] = sqrtf( sxmag * sxmag + symag * symag );
		//sfiltmag[idx] = __fsqrt_rn(sxmag * sxmag + symag * symag );
	}
}	



/*
//NB: this kernel assumes that the magnitude is stored in the x component of the float2 src!
static __global__ void localMinKernel(float *dst, float2 *src, const int M, const int N)
{
	int xidx = threadIdx.x + blockDim.x*blockIdx.x;
	int yidx = threadIdx.y + blockDim.y*blockIdx.y;
	if ( xidx<(N-1) && xidx>=1 && yidx<(M-1) && yidx>=1) //limits so that I don't have to do special junk for the edges, corners
	{
		int idx = yidx*N + xidx;
		float lmin = min(src[idx - N - 1].x, src[idx - 1].x);
		lmin = min(lmin, src[idx + N -1].x);
		lmin = min(lmin, src[idx - N].x);
		lmin = min(lmin, src[idx].x);
		lmin = min(lmin, src[idx + N].x);
		lmin = min(lmin, src[idx - N + 1].x);
		lmin = min(lmin, src[idx + 1].x);
		lmin = min(lmin, src[idx + N +1].x);
		dst[idx] = lmin;
	}
}
*/
//this is terrible, actually. use texture filtering if the 3x3 local neighborhood is actually useful!






static __global__ void computeSIMkernel(float *sim, float *sfiltmag, float *localmag, const float intmax, 
										const int M, const int N)
{
	int xidx = threadIdx.x + blockDim.x*blockIdx.x;
	int yidx = threadIdx.y + blockDim.y*blockIdx.y;
	if (xidx<N && yidx<M) {
		int idx = yidx*N + xidx;
		sim[idx] = sfiltmag[idx] * (intmax - localmag[idx]);
	}
}



static __global__ void addShortKernel(short *dst, const short value, const int M, const int N)
{
	int xidx = threadIdx.x + blockDim.x*blockIdx.x;
	int yidx = threadIdx.y + blockDim.y*blockIdx.y;
	if (xidx<N && yidx<M){
		dst[yidx*N+xidx] += value;
	}
}




//NB: the source and destination are the same location!
//TODO: this might be a good use for atomic operators.
static __global__ void powerKernel(float2 *fsrc, const int xpix, const int ypix,
								   const float pow, const float offset, const float sigma){
	int xidx = threadIdx.x + blockDim.x * blockIdx.x; //pixel index that this thread deals with
	int yidx = threadIdx.y + blockDim.y * blockIdx.y;

	if (xidx < xpix && yidx < ypix){

		//get the FFTW-shifted u,v coordinates (in numbers of samples)
		//option 4) use normalized freqs (just the magnitude) - note that uidx->u here in the nomenclature.
		int halfx = xpix/2, halfy = ypix/2;
		float u = (xidx >= halfx ? (float)(xpix - xidx)/(float)halfx : (float)xidx/(float)halfx);
		float v = (yidx >= halfy ? (float)(ypix - yidx)/(float)halfy : (float)yidx/(float)halfy);

		int idx = yidx * ypix + xidx; //the linear index of the data
		float2 value = fsrc[idx]; //the value to work with

		//do some operation on value using u,v
		//here, i'm using a power filter to keep the lower freqs but kill the high freqs.
		float fkeep = __expf( - ( __powf(u,pow) + __powf(v,pow) )/sigma);
		value.x = value.x * fkeep;
		value.y = value.y * fkeep; 
		//the offset is the fraction to keep no matter what; only (1-offset) of the value
		//is allowed to be changed.

		//store the result back to the destination
		fsrc[idx] = value;
	}
}

static __global__ void fillSfiltxKernel(float2 *sfiltx, const float sigma, 
														   const int xpix, const int ypix)
{
	int xidx = threadIdx.x + blockDim.x * blockIdx.x; //pixel index that this thread deals with
	int yidx = threadIdx.y + blockDim.y * blockIdx.y;

	if (xidx < xpix && yidx < ypix){
		int halfx = xpix/2, halfy = ypix/2;
		float u = (xidx >= halfx ? -(float)(xpix - xidx) : (float)xidx);
		float v = (yidx >= halfy ? -(float)(ypix - yidx) : (float)yidx);

		int idx = yidx * ypix + xidx; //the linear index of the data
		
		//set the value
		sfiltx[idx].y = 0;
		sfiltx[idx].x = -2*u*__expf(-(u*u + v*v)/(2*sigma*sigma));
	}
}



static __global__ void fillSfiltyKernel(float2 *sfiltx, const float sigma, 
														   const int xpix, const int ypix)
{
	int xidx = threadIdx.x + blockDim.x * blockIdx.x; //pixel index that this thread deals with
	int yidx = threadIdx.y + blockDim.y * blockIdx.y;

	if (xidx < xpix && yidx < ypix){
		int halfx = xpix/2, halfy = ypix/2;
		float u = (xidx >= halfx ? -(float)(xpix - xidx) : (float)xidx);
		float v = (yidx >= halfy ? -(float)(ypix - yidx) : (float)yidx);

		int idx = yidx * ypix + xidx; //the linear index of the data
		
		//set the value
		sfiltx[idx].y = 0;
		sfiltx[idx].x = -2*v*__expf(-(u*u + v*v)/(2*sigma*sigma));
	}
}



void reconstruct(float *SIM, short *Sidx, float *Rint, short *Ridx, float *holosrc, 
					 double *z, const int nz, const int M, const int N, 
					 const double lambda, const double pixelsize,
					 const float powOrder, const float powOffset,
					 const float sfiltsigma, const float intmax,
					 float *ratsim, bool rintmode, bool ratsimmode)
{
	float *holodev, *rintdev, *simmaxdev, *sfiltmag, *localmag, *sim, *ratsimdev;
	short *ridxdev, *sidxdev;
	float2 *cholodev, *recdev, *sfiltx, *sfilty, *sx, *sy;
	float adudu, cutoff2, fftscale;
	float2 offsetPhaseExp;
	float sigma;

	dim3 myblock(16, 16);
	dim3 mygrid( roundUpDiv(N,16), roundUpDiv(M,16) );

	mexPrintf("Uploading hologram\n");

	//upload the hologram and convert it to a complex-valued matrix
	hipMalloc( (void**)&holodev, sizeof(float)*M*N);
	hipMalloc( (void**)&cholodev, sizeof(float2)*M*N);
	hipMemcpy( holodev, holosrc, sizeof(float)*M*N, hipMemcpyHostToDevice);
	float_to_complexKernel<<<mygrid, myblock>>>(cholodev, holodev, M, N);
	hipFree(holodev);

	mexPrintf("Creating memory on GPU\n");

	//create variables to track the max SIM, min int
	hipMalloc( (void**)&simmaxdev, sizeof(float)*M*N);
	hipMalloc( (void**)&sidxdev, sizeof(short)*M*N);
	hipMemset(sidxdev, 0, sizeof(short)*M*N);
	//fillFloatKernel<<<mygrid, myblock>>>(rintdev, FLT_MIN, M, N); //don't need this
	if (rintmode){
		hipMalloc( (void**)&rintdev, sizeof(float)*M*N);
		hipMalloc( (void**)&ridxdev, sizeof(short)*M*N);
		hipMemset(ridxdev, 0, sizeof(short)*M*N);
	}

	mexPrintf("Creating FFT plan\n");

	//create an FFT plan
	hipfftHandle plan;
	hipfftPlan2d(&plan, N, M, HIPFFT_C2C);

	mexPrintf("Executing FFT\n");
	
	//compute the (power-modulated) spectra, which will be used for each reconstruction step
	//take the FFT of the hologram data
	hipfftExecC2C(plan, cholodev, cholodev, HIPFFT_FORWARD);

	mexPrintf("Applying power filter\n");

	//apply the power kernel filter
	if (powOrder > 0) {
		//mexPrintf("Applying power kernel\n");
		sigma = -1.0f/log(powOffset);
		powerKernel<<<mygrid, myblock>>>(cholodev, N, M,
			powOrder, powOffset, sigma);
	}

	mexPrintf("Computing steerable filter\n");

	//compute the steerable filter, used for each reconstruction step
	hipMalloc((void**) &sfiltx, sizeof(float2)*M*N);
	hipMalloc((void**) &sfilty, sizeof(float2)*M*N);
	fillSfiltxKernel<<<mygrid,myblock>>>(sfiltx,sfiltsigma,M,N);
	fillSfiltyKernel<<<mygrid,myblock>>>(sfilty,sfiltsigma,M,N);
	hipfftExecC2C(plan, sfiltx, sfiltx, HIPFFT_FORWARD); //compute the freq-domain version of the filter
	hipfftExecC2C(plan, sfilty, sfilty, HIPFFT_FORWARD); 
	//..and reserve memory for the resulting sx, sy gradients
	hipMalloc((void**) &sx, sizeof(float2)*M*N);
	hipMalloc((void**) &sy, sizeof(float2)*M*N);
	hipMalloc((void**) &sfiltmag, sizeof(float)*M*N);



	//reserve memory for the reconstruction
	hipMalloc( (void**)&recdev, sizeof(float2)*M*N);


	//reserve memory for the focus metric components
	hipMalloc( (void**)&localmag, sizeof(float)*M*N);
	hipMalloc( (void**)&sim, sizeof(float)*M*N);
	if (ratsimmode)
		hipMalloc( (void**)&ratsimdev, sizeof(float)*M*N);

	//fillFloatKernel<<<mygrid, myblock>>>(localmag, 1.0f, M, N); //don't need this

	mexPrintf("Computing reconstructions\n");

	//reconstruct the hologram at each depth in z, compare it to the previous
	//minimum intensity
	for (int i=0; i<nz; i++){
		//copy the fft spectrum to a variable that's temporary for each plane
		hipMemcpy(recdev, cholodev, sizeof(float2)*M*N, hipMemcpyDeviceToDevice);

		//set the propagation parameters
		setPropagationParams( (float)(z[i]), (float)lambda, (float)pixelsize, M, N, 
			&adudu, &cutoff2, &offsetPhaseExp, &fftscale, false);

		//apply the propagation kernel (full... could speed this up by ignoring cutoff)
		fresnelKernel<<<mygrid, myblock>>>(recdev, M, N, adudu, 
			cutoff2, offsetPhaseExp, fftscale);

		//take the inverse transform: get an image at the plane.
		hipfftExecC2C(plan, recdev, recdev, HIPFFT_BACKWARD);

		//compute the magnitude
		computeMagKernel<<<mygrid, myblock>>>(recdev, M, N);

		//compute the steerable gradient responses in the x and y directions
		hipMemcpy( sx, recdev, sizeof(float2)*M*N, hipMemcpyDeviceToDevice); //copy the magnitude info
		hipfftExecC2C(plan, sx, sx, HIPFFT_FORWARD); //the spectrum of the magnitude
		hipMemcpy( sy, sx, sizeof(float2)*M*N, hipMemcpyDeviceToDevice); //copy the spectrum information
		//apply the sfilt kernels in the x and y directions
		complexMultKernel<<<mygrid, myblock>>>(sx, sx, sfiltx, M, N);
		complexMultKernel<<<mygrid, myblock>>>(sy, sy, sfilty, M, N);
		hipfftExecC2C(plan, sx, sx, HIPFFT_BACKWARD);
		hipfftExecC2C(plan, sy, sy, HIPFFT_BACKWARD);
		//finally: compute the magnitude and orientation
		sfiltMagKernel<<<mygrid,myblock>>>(sfiltmag, sx, sy, M, N);
		scaleFloatKernel<<<mygrid,myblock>>>(sfiltmag, 1.0f/((float)(M*M) * (float)(N*N)), M, N);
		//TODO: add orientation later, if desired


		//find the local min intensity (for SIM)
		//localMinKernel<<<mygrid, myblock>>>(localmag, recdev, M, N);
		copyRealKernel<<<mygrid, myblock>>>(localmag, recdev, M, N);


		//compute the SIM metric
		computeSIMkernel<<<mygrid, myblock>>>(sim, sfiltmag, localmag, intmax, M, N);
		

		if (rintmode){
			//compare the current intensity to the previous minimum
			if (i==0) 
				copyRealKernel<<<mygrid,myblock>>>(rintdev, recdev, M, N);
			else 
				compareReal2Kernel<<<mygrid, myblock>>>(rintdev, ridxdev, recdev, i, M, N);
		}

		//compare the current SIM to the previous SIMs
		if (i==0) {
			hipMemcpy(simmaxdev, sim, sizeof(float)*M*N, hipMemcpyDeviceToDevice);
			if (ratsimmode)
				hipMemcpy(ratsimdev, rintdev, sizeof(float)*M*N, hipMemcpyDeviceToDevice);
		}
		else {
			if (!ratsimmode)
				compareRealKernel<<<mygrid, myblock>>>(simmaxdev, sidxdev, sim, i, M, N);
			else //ratsimmode==true
				compareRealKernel<<<mygrid, myblock>>>(simmaxdev, sidxdev, sim, ratsimdev, recdev, i, M, N);
		}


	}

	//convert from c-style indexing to matlab-style indexing
	if (rintmode)
		addShortKernel<<<mygrid, myblock>>>(ridxdev, 1, M, N);
	addShortKernel<<<mygrid, myblock>>>(sidxdev, 1, M, N);

	mexPrintf("Copying results back to host\n");

	//return the data to the host
	hipMemcpy(SIM, simmaxdev, sizeof(float)*M*N, hipMemcpyDeviceToHost);
	hipMemcpy(Sidx, sidxdev, sizeof(short)*M*N, hipMemcpyDeviceToHost);
	if (rintmode){
		hipMemcpy(Rint, rintdev, sizeof(float)*M*N, hipMemcpyDeviceToHost);
		hipMemcpy(Ridx, ridxdev, sizeof(short)*M*N, hipMemcpyDeviceToHost);
	}
	if (ratsimmode)
		hipMemcpy(ratsim, ratsimdev, sizeof(float)*M*N, hipMemcpyDeviceToHost);

	mexPrintf("Cleaning up memory and plans\n");

	//clean up!
	hipfftDestroy(plan);
	hipFree(cholodev);
	hipFree(recdev);
	if (rintmode){
		hipFree(rintdev);
		hipFree(ridxdev);
	}
	hipFree(sfiltx);
	hipFree(sfilty);
	hipFree(sx);
	hipFree(sy);
	hipFree(sfiltmag);
	hipFree(sim);
	hipFree(localmag);
	hipFree(sidxdev);
	hipFree(simmaxdev);
	if (ratsimmode)
		hipFree(ratsimdev);
}



// [SIM, Sidx, Rint, Ridx, R_at_SIM] = maxSIMfm(img, z, lambda, pixelsize, ...
//           powerKernOrder, powerKernOffset, sigma, intmax);
void mexFunction( int nlhs, mxArray *plhs[],
                  int nrhs, const mxArray *prhs[])
{
	
	float *holosrc, *Rint, *SIM, *ratsim = NULL;
	double *dsrc;
	unsigned char *isrc;
	short *Ridx, *Sidx;
	double *z, lambda = 658e-9, pixelsize = 9e-6;
	int M, N, nz;
	float powOrder = -1, powOffset = .001, sigma = 2, intmax = 1000;
	bool rintmode, ratsimmode;

	if (nrhs<2)
		mexErrMsgTxt("maxSIMfm(image, z, [lambda], [pixelsize]) requires at least two arguments.");

	//get the size of the hologram
	M = mxGetN(prhs[0]); //number of y-direction pixels
	N = mxGetM(prhs[0]); //number of x-direction pixels
	//NB: I'm mixing M, N so that they align with the Matlab imagesc plots.
	//M is the y-direction , N is the x-direction.
	//this is because Matlab stores its data reading down the rows (row-major), while
	//C is column-wise ordering (column-major)
	
	//mexPrintf("nrhs = %i\n", nrhs);

	//retrieve the depths at which to do reconstructions
	z = mxGetPr(prhs[1]);
	nz = mxGetM(prhs[1])*mxGetN(prhs[1]);
	
	//mexPrintf("M: %i, N: %i, nz: %i\n", M, N, nz);

	//get optional parameters; use defauls otherwise
	if (nrhs>=3)
		if (!mxIsEmpty(prhs[2]))
			lambda = mxGetScalar(prhs[2]);
	if (nrhs>=4)
		if (!mxIsEmpty(prhs[3]))
			pixelsize = mxGetScalar(prhs[3]);
	if (nrhs>=5)
		if (!mxIsEmpty(prhs[4]))
			powOrder = (float) mxGetScalar(prhs[4]);
	if (nrhs>=6)
		if (!mxIsEmpty(prhs[5]))
			powOffset = (float) mxGetScalar(prhs[5]);
	if (nrhs>=7)
		if (!mxIsEmpty(prhs[6]))
			sigma = (float) mxGetScalar(prhs[6]);
	if (nrhs>=8)
		if (!mxIsEmpty(prhs[7]))
			intmax = (float) mxGetScalar(prhs[7]);
			

	//get the holographic image! 
	// note: if it's double or int, need to convert to float.

	mxClassID imgclass = mxGetClassID(prhs[0]);
	switch (imgclass){
		case mxSINGLE_CLASS:
			holosrc = (float*) mxGetPr(prhs[0]);
			break;
		case mxDOUBLE_CLASS:
			dsrc = mxGetPr(prhs[0]);
			holosrc = (float*) mxMalloc( sizeof(float)*M*N );
			double_to_float(holosrc, dsrc, M*N);
			break;
		case mxUINT8_CLASS:
			isrc = (unsigned char*) mxGetPr(prhs[0]);
			holosrc = (float*) mxMalloc( sizeof(float)*M*N );
			uint8_to_float(holosrc, isrc, M*N);
			break;
		default:
			mexErrMsgTxt("Input image needs to be single, double, or uint8.");
			break;
	}
	
	//create outputs
	
	if (nlhs>2)
		rintmode = true;
	else
		rintmode = false;
	if (nlhs>4)
		ratsimmode = true;
	else
		ratsimmode = false;

	plhs[0] = mxCreateNumericMatrix(N, M, mxSINGLE_CLASS, mxREAL); //the order is reversed here
	plhs[1] = mxCreateNumericMatrix(N, M, mxINT16_CLASS, mxREAL); //from what you'd expect... row-major vs column-major.
	SIM = (float*)mxGetPr(plhs[0]);
	Sidx = (short*)mxGetPr(plhs[1]);

	if (rintmode){
		plhs[2] = mxCreateNumericMatrix(N, M, mxSINGLE_CLASS, mxREAL);
		plhs[3] = mxCreateNumericMatrix(N, M, mxINT16_CLASS, mxREAL);
		Rint = (float*)mxGetPr(plhs[2]); //single
		Ridx = (short*)mxGetPr(plhs[3]); //int16
	}

	if (ratsimmode){
		plhs[4] = mxCreateNumericMatrix(N, M, mxSINGLE_CLASS, mxREAL);
		ratsim  = (float*)mxGetPr(plhs[4]);
	}

	//mexPrintf("Derivative: %i\n", derivOrder);

	//reconstruct hologram and find mins
	reconstruct(SIM, Sidx, Rint, Ridx, holosrc, z, nz, M, N, lambda, pixelsize, 
		powOrder, powOffset, sigma, intmax, ratsim, rintmode, ratsimmode);

	//clean up
	if ( imgclass == mxDOUBLE_CLASS || imgclass == mxUINT8_CLASS )
		mxFree(holosrc);
}